
#include <hip/hip_runtime.h>
//========================================================================================
//  Copyright (c) 2018  Pacific Vis Paper Author Submission 1144  -- All rights reserved.
//  See LICENSE.md for details.
//========================================================================================

#define ALL_MASK 1
#define COM_MASK 2
#define ALL_SEL_MASK 4
#define COM_SEL_MASK 8

__device__ __constant__
int lookup[ 31 ] = {
    1,2,4,8,16,32,64,128,256,512,1024,2048,4096,
    8192,16384,32768,65536,131072,262144,524288,
    1048576,2097152,4194304,8388608,16777216,
    33554432,67108864,134217728,268435456,536870912,
    1073741824
};

__device__
bool testSet( int s, int b )
{
    return ( ( b & lookup[ s ] ) && ( b & 1 ) );
}

extern "C" __global__
void divide(
    float * A,
    float * B,
    int N )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx >= N )
    {
        return;
    }

    A[ idx ] /= B[ idx ];
}

extern "C" __global__
void computeTimeSeriesSumsAndCounts(
    float * data,
    int   * bitmasks,
    float * sums,
    float * counts,
    const unsigned long NPT,
    const unsigned long N_STEPS,
    const unsigned int N_SETS,
    const unsigned int V_OFF )
{
    const long int GLOBAL_IDX = blockIdx.x * blockDim.x + threadIdx.x;

    if( GLOBAL_IDX >= NPT )
    {
        return;
    }

    const int pIDX = GLOBAL_IDX / N_STEPS;
    const int tIDX = GLOBAL_IDX % N_STEPS;

    int b = bitmasks[ GLOBAL_IDX ];

    for( int sIDX = 0; sIDX < N_SETS; ++sIDX )
    {
        if( ( ( b & lookup[ sIDX ] ) && ( b & 1 ) ) )
        {
            const float value = data[ V_OFF + pIDX * N_STEPS + tIDX ];
            atomicAdd( & sums[   sIDX * N_STEPS + tIDX ], value );
            atomicAdd( & counts[ sIDX * N_STEPS + tIDX ], 1.0 );
        }
    }
}

extern "C" __global__
void computeCurrentHistograms(
    float * data,
    int   * bitmasks,
    float * ranges,
    float * histograms,
    const unsigned long N_PARTICLES,
    const unsigned long N_STEPS,
    const unsigned long STEP,
    const unsigned int N_VARS,
    const unsigned int N_BINS,
    const unsigned int N_SETS )
{
    const int pIDX = blockIdx.x * blockDim.x + threadIdx.x;

    if( pIDX >= N_PARTICLES )
    {
        return;
    }

    int b = bitmasks[ N_STEPS * pIDX + STEP ];

    for( unsigned int sIDX = 0; sIDX < N_SETS; ++sIDX )
    {
        if( ! testSet( sIDX, b ) )
        {
            continue;
        }
        for( unsigned int vIDX = 0; vIDX < N_VARS; ++vIDX )
        {
            const float MN = ranges[ vIDX*2     ];
            const float MX = ranges[ vIDX*2 + 1 ];
            const float WDTH = MX - MN;

            const float value = data[ vIDX * N_STEPS * N_PARTICLES + pIDX * N_STEPS + STEP ];

            if( value < MN || value > MX )
            {
                continue;
            }

            int bIDX = umin( ( unsigned int ) ( ( value - MN ) / WDTH * N_BINS ), ( unsigned int )( N_BINS - 1 ) );

            atomicAdd( & histograms[ sIDX * N_VARS * N_BINS + vIDX * N_BINS + bIDX ], 1.0 );
        }
    }
}
